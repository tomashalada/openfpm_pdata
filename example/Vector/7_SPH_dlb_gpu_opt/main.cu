
#include <hip/hip_runtime.h>
/*! \page Vector_7_sph_dlb_gpu_opt Vector 7 SPH Dam break simulation with Dynamic load balacing on Multi-GPU (optimized version)
 *
 *
 * [TOC]
 *
 *
 * # SPH with Dynamic load Balancing on GPU (Optimized) # {#SPH_dlb_gpu_opt}
 *
 *
 * This example show the classical SPH Dam break simulation with load balancing and dynamic load balancing. The main difference with
 * \ref{SPH_dlb} is that here we use GPU and 1.2 Millions particles. Simulate 1.5 second should be duable on a 1050Ti within a couple
 * of hours.
 *
 * \htmlonly
 * <a href="#" onclick="hide_show('vector-video-3')" >Simulation video 1</a><br>
 * <div style="display:none" id="vector-video-3">
 * <video id="vid3" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu1.mp4" type="video/mp4"></video>
 * </div>
 * <a href="#" onclick="hide_show('vector-video-4')" >Simulation video 2</a><br>
 * <div style="display:none" id="vector-video-4">
 * <video id="vid4" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu2.mp4" type="video/mp4"></video>
 * </div>
 * <a href="#" onclick="hide_show('vector-video-15')" >Simulation video 3</a><br>
 * <div style="display:none" id="vector-video-15">
 * <video id="vid15" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu3.mp4" type="video/mp4"></video>
 * </div>
 * \endhtmlonly
 *
 *
 * ## GPU ## {#e7_sph_inclusion}
 *
 * This example is an optimization of the example \ref SPH_dlb_gpu all the optimization operated on this example has been explained
 * here \ref e3_md_gpu_opt so we will not go into the details
 *
 * we report the full code here
 *
 *
 */

#ifdef __NVCC__

#include "Vector/vector_dist.hpp"
#include <math.h>
#include "Draw/DrawParticles.hpp"

typedef float real_number;

// A constant to indicate boundary particles
#define BOUNDARY 0

// A constant to indicate fluid particles
#define FLUID 1

// initial spacing between particles dp in the formulas
const real_number dp = 0.00425;
// Maximum height of the fluid water
// is going to be calculated and filled later on
real_number h_swl = 0.0;

// c_s in the formulas (constant used to calculate the sound speed)
const real_number coeff_sound = 20.0;

// gamma in the formulas
const real_number gamma_ = 7.0;

// sqrt(3.0*dp*dp) support of the kernel
const real_number H = 0.00736121593217;

// Eta in the formulas
const real_number Eta2 = 0.01 * H*H;

const real_number FourH2 = 4.0 * H*H;

// alpha in the formula
const real_number visco = 0.1;

// cbar in the formula (calculated later)
real_number cbar = 0.0;

// Mass of the fluid particles
const real_number MassFluid = 0.0000767656;

// Mass of the boundary particles
const real_number MassBound = 0.0000767656;

//

// End simulation time
#ifdef TEST_RUN
const real_number t_end = 0.001;
#else
const real_number t_end = 1.50;
#endif

// Gravity acceleration
const real_number gravity = 9.81;

// Reference densitu 1000Kg/m^3
const real_number rho_zero = 1000.0;

// Filled later require h_swl, it is b in the formulas
real_number B = 0.0;

// Constant used to define time integration
const real_number CFLnumber = 0.2;

// Minimum T
const real_number DtMin = 0.00001;

// Minimum Rho allowed
const real_number RhoMin = 700.0;

// Maximum Rho allowed
const real_number RhoMax = 1300.0;

// Filled in initialization
real_number max_fluid_height = 0.0;

// Properties

// FLUID or BOUNDARY
const size_t type = 0;

// Density
const int rho = 1;

// Density at step n-1
const int rho_prev = 2;

// Pressure
const int Pressure = 3;

// Delta rho calculated in the force calculation
const int drho = 4;

// calculated force
const int force = 5;

// velocity
const int velocity = 6;

// velocity at previous step
const int velocity_prev = 7;

const int red = 8;

const int red2 = 9;

// Type of the vector containing particles
typedef vector_dist_gpu<3,real_number,aggregate<unsigned int,real_number,  real_number,    real_number,     real_number,     real_number[3], real_number[3], real_number[3], real_number, real_number>> particles;
//                                              |          |             |               |                |                |               |               |               |            |
//                                              |          |             |               |                |                |               |               |               |            |
//                                             type      density       density        Pressure          delta            force          velocity        velocity        reduction     another
//                                                                     at n-1                           density                                         at n - 1        buffer        reduction buffer


struct ModelCustom
{
	template<typename Decomposition, typename vector> inline void addComputation(Decomposition & dec,
			                                                                     vector & vd,
																				 size_t v,
																				 size_t p)
	{
		if (vd.template getProp<type>(p) == FLUID)
			dec.addComputationCost(v,4);
		else
			dec.addComputationCost(v,3);
	}

	template<typename Decomposition> inline void applyModel(Decomposition & dec, size_t v)
	{
		dec.setSubSubDomainComputationCost(v, dec.getSubSubDomainComputationCost(v) * dec.getSubSubDomainComputationCost(v));
	}

	real_number distributionTol()
	{
		return 1.01;
	}
};

template<typename vd_type>
__global__ void EqState_gpu(vd_type vd, real_number B)
{
	auto a = GET_PARTICLE(vd);

	real_number rho_a = vd.template getProp<rho>(a);
	real_number rho_frac = rho_a / rho_zero;

	vd.template getProp<Pressure>(a) = B*( rho_frac*rho_frac*rho_frac*rho_frac*rho_frac*rho_frac*rho_frac - 1.0);
}

inline void EqState(particles & vd)
{
	auto it = vd.getDomainIteratorGPU();

	EqState_gpu<<<it.wthr,it.thr>>>(vd.toKernel(),B);
}


const real_number a2 = 1.0/M_PI/H/H/H;

inline __device__ __host__ real_number Wab(real_number r)
{
	r /= H;

	if (r < 1.0)
		return (1.0 - 3.0/2.0*r*r + 3.0/4.0*r*r*r)*a2;
	else if (r < 2.0)
		return (1.0/4.0*(2.0 - r*r)*(2.0 - r*r)*(2.0 - r*r))*a2;
	else
		return 0.0;
}


const real_number c1 = -3.0/M_PI/H/H/H/H;
const real_number d1 = 9.0/4.0/M_PI/H/H/H/H;
const real_number c2 = -3.0/4.0/M_PI/H/H/H/H;
const real_number a2_4 = 0.25*a2;
// Filled later
real_number W_dap = 0.0;

inline __device__ __host__ void DWab(Point<3,real_number> & dx, Point<3,real_number> & DW, real_number r)
{
	const real_number qq=r/H;

    real_number qq2 = qq * qq;
    real_number fac1 = (c1*qq + d1*qq2)/r;
    real_number b1 = (qq < 1.0)?1.0f:0.0f;

    real_number wqq = (2.0 - qq);
    real_number fac2 = c2 * wqq * wqq / r;
    real_number b2 = (qq >= 1.0 && qq < 2.0)?1.0f:0.0f;

    real_number factor = (b1*fac1 + b2*fac2);

    DW.get(0) = factor * dx.get(0);
    DW.get(1) = factor * dx.get(1);
    DW.get(2) = factor * dx.get(2);
}

// Tensile correction
inline __device__ __host__  real_number Tensile(real_number r, real_number rhoa, real_number rhob, real_number prs1, real_number prs2, real_number W_dap)
{
	const real_number qq=r/H;
	//-Cubic Spline kernel
	real_number wab;
	if(r>H)
	{
		real_number wqq1=2.0f-qq;
		real_number wqq2=wqq1*wqq1;

		wab=a2_4*(wqq2*wqq1);
	}
	else
	{
	    real_number wqq2=qq*qq;
	    real_number wqq3=wqq2*qq;

	    wab=a2*(1.0f-1.5f*wqq2+0.75f*wqq3);
	}

	//-Tensile correction.
	real_number fab=wab*W_dap;
	fab*=fab; fab*=fab; //fab=fab^4
	const real_number tensilp1=(prs1/(rhoa*rhoa))*(prs1>0? 0.01: -0.2);
	const real_number tensilp2=(prs2/(rhob*rhob))*(prs2>0? 0.01: -0.2);

	return (fab*(tensilp1+tensilp2));
}


inline __device__ __host__ real_number Pi(const Point<3,real_number> & dr, real_number rr2, Point<3,real_number> & dv, real_number rhoa, real_number rhob, real_number massb, real_number cbar, real_number & visc)
{
	const real_number dot = dr.get(0)*dv.get(0) + dr.get(1)*dv.get(1) + dr.get(2)*dv.get(2);
	const real_number dot_rr2 = dot/(rr2+Eta2);
	visc=(dot_rr2 < visc)?visc:dot_rr2;

	if(dot < 0)
	{
		const float amubar=H*dot_rr2;
		const float robar=(rhoa+rhob)*0.5f;
		const float pi_visc=(-visco*cbar*amubar/robar);

		return pi_visc;
    }
	else
		return 0.0;
}

template<typename particles_type, typename NN_type>
__global__ void calc_forces_gpu(particles_type vd, NN_type NN, real_number W_dap, real_number cbar)
{
	// ... a
	unsigned int a;
	GET_PARTICLE_SORT(a,NN);

	real_number max_visc = 0.0;

	// Get the position xp of the particle
	Point<3,real_number> xa = vd.getPos(a);

	// Type of the particle
	unsigned int typea = vd.getProp<type>(a);

	// Take the mass of the particle dependently if it is FLUID or BOUNDARY
	real_number massa = (vd.getProp<type>(a) == FLUID)?MassFluid:MassBound;

	// Get the density of the of the particle a
	real_number rhoa = vd.getProp<rho>(a);

	// Get the pressure of the particle a
	real_number Pa = vd.getProp<Pressure>(a);

	// Get the Velocity of the particle a
	Point<3,real_number> va = vd.getProp<velocity>(a);

	Point<3,real_number> force_;
	force_.get(0) = 0.0;
	force_.get(1) = 0.0;
	force_.get(2) = -gravity;
	real_number drho_ = 0.0;

	// Get an iterator over the neighborhood particles of p
	auto Np = NN.getNNIteratorBox(NN.getCell(xa));

	// For each neighborhood particle
	while (Np.isNext() == true)
	{
		// ... q
		auto b = Np.get_sort();

		// Get the position xp of the particle
		Point<3,real_number> xb = vd.getPos(b);

		// if (p == q) skip this particle
		if (a == b)	{++Np; continue;};

        unsigned int typeb = vd.getProp<type>(b);

        real_number massb = (typeb == FLUID)?MassFluid:MassBound;
        Point<3,real_number> vb = vd.getProp<velocity>(b);
        real_number Pb = vd.getProp<Pressure>(b);
        real_number rhob = vd.getProp<rho>(b);

		// Get the distance between p and q
		Point<3,real_number> dr = xa - xb;
		// take the norm of this vector
		real_number r2 = norm2(dr);

		// if they interact
		if (r2 < FourH2)
		{
			real_number r = sqrt(r2);

			Point<3,real_number> v_rel = va - vb;

			Point<3,real_number> DW;
			DWab(dr,DW,r);

			real_number factor = - massb*((vd.getProp<Pressure>(a) + vd.getProp<Pressure>(b)) / (rhoa * rhob) + Tensile(r,rhoa,rhob,Pa,Pb,W_dap) + Pi(dr,r2,v_rel,rhoa,rhob,massb,cbar,max_visc));

			// Bound - Bound does not produce any change
			factor = (typea == BOUNDARY && typeb == BOUNDARY)?0.0f:factor;

			force_.get(0) += factor * DW.get(0);
			force_.get(1) += factor * DW.get(1);
			force_.get(2) += factor * DW.get(2);

			real_number scal = massb*(v_rel.get(0)*DW.get(0)+v_rel.get(1)*DW.get(1)+v_rel.get(2)*DW.get(2));
			scal = (typea == BOUNDARY && typeb == BOUNDARY)?0.0f:scal;

			drho_ += scal;
		}

		++Np;
	}

	vd.getProp<red>(a) = max_visc;

	vd.template getProp<force>(a)[0] = force_.get(0);
	vd.template getProp<force>(a)[1] = force_.get(1);
	vd.template getProp<force>(a)[2] = force_.get(2);
	vd.template getProp<drho>(a) = drho_;
}

template<typename CellList> inline void calc_forces(particles & vd, CellList & NN, real_number & max_visc, size_t cnt)
{
	auto part = vd.getDomainIteratorGPU(32);

	// Update the cell-list
	vd.updateCellList(NN);

	calc_forces_gpu<<<part.wthr,part.thr>>>(vd.toKernel_sorted(),NN.toKernel(),W_dap,cbar);

	vd.merge_sort<force,drho,red>(NN);

	max_visc = reduce_local<red,_max_>(vd);
}

template<typename vector_type>
__global__ void max_acceleration_and_velocity_gpu(vector_type vd)
{
	auto a = GET_PARTICLE(vd);

	Point<3,real_number> acc(vd.getProp<force>(a));
	vd.getProp<red>(a) = norm(acc);

	Point<3,real_number> vel(vd.getProp<velocity>(a));
	vd.getProp<red2>(a) = norm(vel);
}

void max_acceleration_and_velocity(particles & vd, real_number & max_acc, real_number & max_vel)
{
	// Calculate the maximum acceleration
	auto part = vd.getDomainIteratorGPU();

	max_acceleration_and_velocity_gpu<<<part.wthr,part.thr>>>(vd.toKernel());

	max_acc = reduce_local<red,_max_>(vd);
	max_vel = reduce_local<red2,_max_>(vd);

	Vcluster<> & v_cl = create_vcluster();
	v_cl.max(max_acc);
	v_cl.max(max_vel);
	v_cl.execute();
}


real_number calc_deltaT(particles & vd, real_number ViscDtMax)
{
	real_number Maxacc = 0.0;
	real_number Maxvel = 0.0;
	max_acceleration_and_velocity(vd,Maxacc,Maxvel);

	//-dt1 depends on force per unit mass.
	const real_number dt_f = (Maxacc)?sqrt(H/Maxacc):std::numeric_limits<float>::max();

	//-dt2 combines the Courant and the viscous time-step controls.
	const real_number dt_cv = H/(std::max(cbar,Maxvel*10.f) + H*ViscDtMax);

	//-dt new value of time step.
	real_number dt=real_number(CFLnumber)*std::min(dt_f,dt_cv);
	if(dt<real_number(DtMin))
	{dt=real_number(DtMin);}

	return dt;
}

template<typename vector_dist_type>
__global__ void verlet_int_gpu(vector_dist_type vd, real_number dt, real_number dt2, real_number dt205)
{
	// ... a
	auto a = GET_PARTICLE(vd);

	// if the particle is boundary
	if (vd.template getProp<type>(a) == BOUNDARY)
	{
		// Update rho
		real_number rhop = vd.template getProp<rho>(a);

		// Update only the density
    	vd.template getProp<velocity>(a)[0] = 0.0;
    	vd.template getProp<velocity>(a)[1] = 0.0;
    	vd.template getProp<velocity>(a)[2] = 0.0;
    	real_number rhonew = vd.template getProp<rho_prev>(a) + dt2*vd.template getProp<drho>(a);
    	vd.template getProp<rho>(a) = (rhonew < rho_zero)?rho_zero:rhonew;

	    vd.template getProp<rho_prev>(a) = rhop;

	    vd.template getProp<red>(a) = 0;

		return;
	}

	//-Calculate displacement and update position / Calcula desplazamiento y actualiza posicion.
	real_number dx = vd.template getProp<velocity>(a)[0]*dt + vd.template getProp<force>(a)[0]*dt205;
    real_number dy = vd.template getProp<velocity>(a)[1]*dt + vd.template getProp<force>(a)[1]*dt205;
    real_number dz = vd.template getProp<velocity>(a)[2]*dt + vd.template getProp<force>(a)[2]*dt205;

    vd.getPos(a)[0] += dx;
    vd.getPos(a)[1] += dy;
    vd.getPos(a)[2] += dz;

    real_number velX = vd.template getProp<velocity>(a)[0];
    real_number velY = vd.template getProp<velocity>(a)[1];
    real_number velZ = vd.template getProp<velocity>(a)[2];

    real_number rhop = vd.template getProp<rho>(a);

	vd.template getProp<velocity>(a)[0] = vd.template getProp<velocity_prev>(a)[0] + vd.template getProp<force>(a)[0]*dt2;
	vd.template getProp<velocity>(a)[1] = vd.template getProp<velocity_prev>(a)[1] + vd.template getProp<force>(a)[1]*dt2;
	vd.template getProp<velocity>(a)[2] = vd.template getProp<velocity_prev>(a)[2] + vd.template getProp<force>(a)[2]*dt2;
	vd.template getProp<rho>(a) = vd.template getProp<rho_prev>(a) + dt2*vd.template getProp<drho>(a);

    // Check if the particle go out of range in space and in density
    if (vd.getPos(a)[0] <  0.0 || vd.getPos(a)[1] < 0.0 || vd.getPos(a)[2] < 0.0 ||
        vd.getPos(a)[0] >  1.61 || vd.getPos(a)[1] > 0.68 || vd.getPos(a)[2] > 0.50 ||
		vd.template getProp<rho>(a) < RhoMin || vd.template getProp<rho>(a) > RhoMax)
    {vd.template getProp<red>(a) = 1;}
    else
    {vd.template getProp<red>(a) = 0;}


    vd.template getProp<velocity_prev>(a)[0] = velX;
    vd.template getProp<velocity_prev>(a)[1] = velY;
    vd.template getProp<velocity_prev>(a)[2] = velZ;
    vd.template getProp<rho_prev>(a) = rhop;
}

size_t cnt = 0;

void verlet_int(particles & vd, real_number dt)
{
	// particle iterator
	auto part = vd.getDomainIteratorGPU();

	real_number dt205 = dt*dt*0.5;
	real_number dt2 = dt*2.0;

	verlet_int_gpu<<<part.wthr,part.thr>>>(vd.toKernel(),dt,dt2,dt205);

	// remove the particles marked
	remove_marked<red>(vd);

	// increment the iteration counter
	cnt++;
}

template<typename vector_type>
__global__ void euler_int_gpu(vector_type vd,real_number dt, real_number dt205)
{
	// ... a
	auto a = GET_PARTICLE(vd);

	// if the particle is boundary
	if (vd.template getProp<type>(a) == BOUNDARY)
	{
		// Update rho
		real_number rhop = vd.template getProp<rho>(a);

		// Update only the density
    	vd.template getProp<velocity>(a)[0] = 0.0;
    	vd.template getProp<velocity>(a)[1] = 0.0;
    	vd.template getProp<velocity>(a)[2] = 0.0;
    	real_number rhonew = vd.template getProp<rho>(a) + dt*vd.template getProp<drho>(a);
    	vd.template getProp<rho>(a) = (rhonew < rho_zero)?rho_zero:rhonew;

	    vd.template getProp<rho_prev>(a) = rhop;

	    vd.template getProp<red>(a) = 0;

		return;
	}

	//-Calculate displacement and update position / Calcula desplazamiento y actualiza posicion.
	real_number dx = vd.template getProp<velocity>(a)[0]*dt + vd.template getProp<force>(a)[0]*dt205;
    real_number dy = vd.template getProp<velocity>(a)[1]*dt + vd.template getProp<force>(a)[1]*dt205;
    real_number dz = vd.template getProp<velocity>(a)[2]*dt + vd.template getProp<force>(a)[2]*dt205;

    vd.getPos(a)[0] += dx;
    vd.getPos(a)[1] += dy;
    vd.getPos(a)[2] += dz;

    real_number velX = vd.template getProp<velocity>(a)[0];
    real_number velY = vd.template getProp<velocity>(a)[1];
    real_number velZ = vd.template getProp<velocity>(a)[2];
    real_number rhop = vd.template getProp<rho>(a);

	vd.template getProp<velocity>(a)[0] = vd.template getProp<velocity>(a)[0] + vd.template getProp<force>(a)[0]*dt;
	vd.template getProp<velocity>(a)[1] = vd.template getProp<velocity>(a)[1] + vd.template getProp<force>(a)[1]*dt;
   	vd.template getProp<velocity>(a)[2] = vd.template getProp<velocity>(a)[2] + vd.template getProp<force>(a)[2]*dt;
   	vd.template getProp<rho>(a) = vd.template getProp<rho>(a) + dt*vd.template getProp<drho>(a);

    // Check if the particle go out of range in space and in density
    if (vd.getPos(a)[0] <  0.0 || vd.getPos(a)[1] < 0.0 || vd.getPos(a)[2] < 0.0 ||
        vd.getPos(a)[0] >  1.61 || vd.getPos(a)[1] > 0.68 || vd.getPos(a)[2] > 0.50 ||
		vd.template getProp<rho>(a) < RhoMin || vd.template getProp<rho>(a) > RhoMax)
    {vd.template getProp<red>(a) = 1;}
    else
    {vd.template getProp<red>(a) = 0;}

    vd.template getProp<velocity_prev>(a)[0] = velX;
    vd.template getProp<velocity_prev>(a)[1] = velY;
    vd.template getProp<velocity_prev>(a)[2] = velZ;
    vd.template getProp<rho_prev>(a) = rhop;
}

void euler_int(particles & vd, real_number dt)
{

	// particle iterator
	auto part = vd.getDomainIteratorGPU();

	real_number dt205 = dt*dt*0.5;

	euler_int_gpu<<<part.wthr,part.thr>>>(vd.toKernel(),dt,dt205);

	// remove the particles
	remove_marked<red>(vd);

	cnt++;
}

template<typename vector_type, typename NN_type>
__global__ void sensor_pressure_gpu(vector_type vd, NN_type NN, Point<3,real_number> probe, real_number * press_tmp)
{
	real_number tot_ker = 0.0;

	// Get the position of the probe i
	Point<3,real_number> xp = probe;

	// get the iterator over the neighbohood particles of the probes position
	auto itg = NN.getNNIteratorBox(NN.getCell(xp));
	while (itg.isNext())
	{
		auto q = itg.get_sort();

		// Only the fluid particles are importants
		if (vd.template getProp<type>(q) != FLUID)
		{
			++itg;
			continue;
		}

		// Get the position of the neighborhood particle q
		Point<3,real_number> xq = vd.getPos(q);

		// Calculate the contribution of the particle to the pressure
		// of the probe
		real_number r = sqrt(norm2(xp - xq));

		real_number ker = Wab(r) * (MassFluid / rho_zero);

		// Also keep track of the calculation of the summed
		// kernel
		tot_ker += ker;

		// Add the total pressure contribution
		*press_tmp += vd.template getProp<Pressure>(q) * ker;

		// next neighborhood particle
		++itg;
	}

	// We calculate the pressure normalizing the
	// sum over all kernels
	if (tot_ker == 0.0)
	{*press_tmp = 0.0;}
	else
	{*press_tmp = 1.0 / tot_ker * *press_tmp;}
}

template<typename Vector, typename CellList>
inline void sensor_pressure(Vector & vd,
                            CellList & NN,
                            openfpm::vector<openfpm::vector<real_number>> & press_t,
                            openfpm::vector<Point<3,real_number>> & probes)
{
    Vcluster<> & v_cl = create_vcluster();

    press_t.add();

    for (size_t i = 0 ; i < probes.size() ; i++)
    {
    	// A float variable to calculate the pressure of the problem
    	CudaMemory press_tmp_(sizeof(real_number));
    	real_number press_tmp;

        // if the probe is inside the processor domain
		if (vd.getDecomposition().isLocal(probes.get(i)) == true)
		{
			sensor_pressure_gpu<<<1,1>>>(vd.toKernel_sorted(),NN.toKernel(),probes.get(i),(real_number *)press_tmp_.toKernel());

			vd.merge<Pressure>(NN);

			// move calculated pressure on
			press_tmp_.deviceToHost();
			press_tmp = *(real_number *)press_tmp_.getPointer();
		}

		// This is not necessary in principle, but if you
		// want to make all processor aware of the history of the calculated
		// pressure we have to execute this
		v_cl.sum(press_tmp);
		v_cl.execute();

		// We add the calculated pressure into the history
		press_t.last().add(press_tmp);
	}
}

int main(int argc, char* argv[])
{
    // initialize the library
	openfpm_init(&argc,&argv);

	// It contain for each time-step the value detected by the probes
	openfpm::vector<openfpm::vector<real_number>> press_t;
	openfpm::vector<Point<3,real_number>> probes;

	probes.add({0.8779,0.3,0.02});
	probes.add({0.754,0.31,0.02});

	// Here we define our domain a 2D box with internals from 0 to 1.0 for x and y
	Box<3,real_number> domain({-0.05,-0.05,-0.05},{1.7010,0.7065,0.511});
	size_t sz[3] = {413,179,133};

	// Fill W_dap
	W_dap = 1.0/Wab(H/1.5);

	// Here we define the boundary conditions of our problem
    size_t bc[3]={NON_PERIODIC,NON_PERIODIC,NON_PERIODIC};

	// extended boundary around the domain, and the processor domain
	Ghost<3,real_number> g(2*H);

	particles vd(0,domain,bc,g,DEC_GRAN(128));

	//! \cond [draw fluid] \endcond

	// You can ignore all these dp/2.0 is a trick to reach the same initialization
	// of Dual-SPH that use a different criteria to draw particles
	Box<3,real_number> fluid_box({dp/2.0,dp/2.0,dp/2.0},{0.4+dp/2.0,0.67-dp/2.0,0.3+dp/2.0});

	// return an iterator to the fluid particles to add to vd
	auto fluid_it = DrawParticles::DrawBox(vd,sz,domain,fluid_box);

	// here we fill some of the constants needed by the simulation
	max_fluid_height = fluid_it.getBoxMargins().getHigh(2);
	h_swl = fluid_it.getBoxMargins().getHigh(2) - fluid_it.getBoxMargins().getLow(2);
	B = (coeff_sound)*(coeff_sound)*gravity*h_swl*rho_zero / gamma_;
	cbar = coeff_sound * sqrt(gravity * h_swl);

	// for each particle inside the fluid box ...
	while (fluid_it.isNext())
	{
		// ... add a particle ...
		vd.add();

		// ... and set it position ...
		vd.getLastPos()[0] = fluid_it.get().get(0);
		vd.getLastPos()[1] = fluid_it.get().get(1);
		vd.getLastPos()[2] = fluid_it.get().get(2);

		// and its type.
		vd.template getLastProp<type>() = FLUID;

		// We also initialize the density of the particle and the hydro-static pressure given by
		//
		// rho_zero*g*h = P
		//
		// rho_p = (P/B + 1)^(1/Gamma) * rho_zero
		//

		vd.template getLastProp<Pressure>() = rho_zero * gravity *  (max_fluid_height - fluid_it.get().get(2));

		vd.template getLastProp<rho>() = pow(vd.template getLastProp<Pressure>() / B + 1, 1.0/gamma_) * rho_zero;
		vd.template getLastProp<rho_prev>() = vd.template getLastProp<rho>();
		vd.template getLastProp<velocity>()[0] = 0.0;
		vd.template getLastProp<velocity>()[1] = 0.0;
		vd.template getLastProp<velocity>()[2] = 0.0;

		vd.template getLastProp<velocity_prev>()[0] = 0.0;
		vd.template getLastProp<velocity_prev>()[1] = 0.0;
		vd.template getLastProp<velocity_prev>()[2] = 0.0;

		// next fluid particle
		++fluid_it;
	}

	// Recipient
	Box<3,real_number> recipient1({0.0,0.0,0.0},{1.6+dp/2.0,0.67+dp/2.0,0.4+dp/2.0});
	Box<3,real_number> recipient2({dp,dp,dp},{1.6-dp/2.0,0.67-dp/2.0,0.4+dp/2.0});

	Box<3,real_number> obstacle1({0.9,0.24-dp/2.0,0.0},{1.02+dp/2.0,0.36,0.45+dp/2.0});
	Box<3,real_number> obstacle2({0.9+dp,0.24+dp/2.0,0.0},{1.02-dp/2.0,0.36-dp,0.45-dp/2.0});
	Box<3,real_number> obstacle3({0.9+dp,0.24,0.0},{1.02,0.36,0.45});

	openfpm::vector<Box<3,real_number>> holes;
	holes.add(recipient2);
	holes.add(obstacle1);
	auto bound_box = DrawParticles::DrawSkin(vd,sz,domain,holes,recipient1);

	while (bound_box.isNext())
	{
		vd.add();

		vd.getLastPos()[0] = bound_box.get().get(0);
		vd.getLastPos()[1] = bound_box.get().get(1);
		vd.getLastPos()[2] = bound_box.get().get(2);

		vd.template getLastProp<type>() = BOUNDARY;
		vd.template getLastProp<rho>() = rho_zero;
		vd.template getLastProp<rho_prev>() = rho_zero;
		vd.template getLastProp<velocity>()[0] = 0.0;
		vd.template getLastProp<velocity>()[1] = 0.0;
		vd.template getLastProp<velocity>()[2] = 0.0;

		vd.template getLastProp<velocity_prev>()[0] = 0.0;
		vd.template getLastProp<velocity_prev>()[1] = 0.0;
		vd.template getLastProp<velocity_prev>()[2] = 0.0;

		++bound_box;
	}

	auto obstacle_box = DrawParticles::DrawSkin(vd,sz,domain,obstacle2,obstacle1);

	while (obstacle_box.isNext())
	{
		vd.add();

		vd.getLastPos()[0] = obstacle_box.get().get(0);
		vd.getLastPos()[1] = obstacle_box.get().get(1);
		vd.getLastPos()[2] = obstacle_box.get().get(2);

		vd.template getLastProp<type>() = BOUNDARY;
		vd.template getLastProp<rho>() = rho_zero;
		vd.template getLastProp<rho_prev>() = rho_zero;
		vd.template getLastProp<velocity>()[0] = 0.0;
		vd.template getLastProp<velocity>()[1] = 0.0;
		vd.template getLastProp<velocity>()[2] = 0.0;

		vd.template getLastProp<velocity_prev>()[0] = 0.0;
		vd.template getLastProp<velocity_prev>()[1] = 0.0;
		vd.template getLastProp<velocity_prev>()[2] = 0.0;

		++obstacle_box;
	}

	vd.map();

	// Now that we fill the vector with particles
	ModelCustom md;

	vd.addComputationCosts(md);
	vd.getDecomposition().decompose();
	vd.map();

	///////////////////////////

	// Ok the initialization is done on CPU on GPU we are doing the main loop, so first we offload all properties on GPU

	vd.hostToDevicePos();
	vd.template hostToDeviceProp<type,rho,rho_prev,Pressure,velocity>();

	vd.ghost_get<type,rho,Pressure,velocity>(RUN_ON_DEVICE);

	auto NN = vd.getCellListGPU(2*H / 2.0);

	timer tot_sim;
	tot_sim.start();

	size_t write = 0;
	size_t it = 0;
	size_t it_reb = 0;
	real_number t = 0.0;
	while (t <= t_end)
	{
		Vcluster<> & v_cl = create_vcluster();
		timer it_time;

		////// Do rebalancing every 200 timesteps
		it_reb++;
		if (it_reb == 300)
		{
			vd.map(RUN_ON_DEVICE);

			// Rebalancer for now work on CPU , so move to CPU
            vd.deviceToHostPos();
            vd.template deviceToHostProp<type>();

			it_reb = 0;
			ModelCustom md;
			vd.addComputationCosts(md);
			vd.getDecomposition().decompose();

			if (v_cl.getProcessUnitID() == 0)
			{std::cout << "REBALANCED " << it_reb << std::endl;}
		}

		vd.map(RUN_ON_DEVICE);

		// Calculate pressure from the density
		EqState(vd);

		real_number max_visc = 0.0;

		vd.ghost_get<type,rho,Pressure,velocity>(RUN_ON_DEVICE);

		// Calc forces
		calc_forces(vd,NN,max_visc,cnt);

		// Get the maximum viscosity term across processors
		v_cl.max(max_visc);
		v_cl.execute();

		// Calculate delta t integration
		real_number dt = calc_deltaT(vd,max_visc);

		// VerletStep or euler step
		it++;
		if (it < 40)
			verlet_int(vd,dt);
		else
		{
			euler_int(vd,dt);
			it = 0;
		}

		t += dt;

		if (write < t*100)
		{
			// Sensor pressure require update ghost, so we ensure that particles are distributed correctly
			// and ghost are updated
			vd.map(RUN_ON_DEVICE);
			vd.ghost_get<type,rho,Pressure,velocity>(RUN_ON_DEVICE);
			vd.updateCellList(NN);

			// calculate the pressure at the sensor points
			//sensor_pressure(vd,NN,press_t,probes);

			std::cout << "OUTPUT " << dt << std::endl;

			// When we write we have move all the particles information back to CPU

			vd.deviceToHostPos();
			vd.deviceToHostProp<type,rho,rho_prev,Pressure,drho,force,velocity,velocity_prev,red,red2>();

			vd.write_frame("Geometry",write,VTK_WRITER | FORMAT_BINARY);
			write++;

			if (v_cl.getProcessUnitID() == 0)
			{std::cout << "TIME: " << t << "  write " << it_time.getwct() << "   " << it_reb << "   " << cnt << " Max visc: " << max_visc << "   " << vd.size_local()  << std::endl;}
		}
		else
		{
			if (v_cl.getProcessUnitID() == 0)
			{std::cout << "TIME: " << t << "  " << it_time.getwct() << "   " << it_reb << "   " << cnt  << " Max visc: " << max_visc << "   " << vd.size_local() << std::endl;}
		}
	}

	tot_sim.stop();
	std::cout << "Time to complete: " << tot_sim.getwct() << " seconds" << std::endl;

	openfpm_finalize();
}
 
#else

int main(int argc, char* argv[])
{
        return 0;
}

#endif
