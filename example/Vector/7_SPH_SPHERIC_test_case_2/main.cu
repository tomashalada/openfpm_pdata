
#include <hip/hip_runtime.h>
/*! \page Vector_7_sph_dlb_gpu_more_opt Vector 7 SPH Dam break simulation with Dynamic load balacing on Multi-GPU (more
 * optimized version)
 *
 *
 */

#ifdef __NVCC__

   #define PRINT_STACKTRACE
   #define STOP_ON_ERROR
   #define OPENMPI
   #define SCAN_WITH_CUB
   #define SORT_WITH_CUB
//#define SE_CLASS1

//#define USE_LOW_REGISTER_ITERATOR

   #include "Vector/vector_dist.hpp"
   #include <math.h>
   #include "Draw/DrawParticles.hpp"

   #include "defs.h"
typedef float real_number;

   // A constant to indicate boundary particles
   #define BOUNDARY 0

   // A constant to indicate fluid particles
   #define FLUID 1

// initial spacing between particles dp in the formulas
const real_number dp = 0.02;
// Maximum height of the fluid water
// is going to be calculated and filled later on
real_number h_swl = 0.0;

// c_s in the formulas (constant used to calculate the sound speed)
const real_number coeff_sound = 20.0;

// gamma in the formulas
const real_number gamma_ = 7.0;

// Hcoef = sqrt(3) or Hcoef = 2
const real_number Hcoef = 2.0;
// Hcoef * dp support of the kernel
const real_number H = Hcoef * dp;

// Eta in the formulas
const real_number Eta2 = 0.01 * H * H;

const real_number FourH2 = 4.0 * H * H;

// alpha in the formula
const real_number visco = 0.1;

// Reference densitu 1000Kg/m^3
const real_number rho_zero = 1000.0;

// cbar in the formula (calculated later)
real_number cbar = 0.0;

// Mass of the fluid particles
const real_number MassFluid = rho_zero * dp * dp * dp;

// Mass of the boundary particles
const real_number MassBound = rho_zero * dp * dp * dp;

// End simulation time
//#ifdef TEST_RUN
//const real_number simulatioEndTime = 0.001;
//#else
const real_number simulatioEndTime = 1.0;
//#endif

// Gravity acceleration
const real_number gravity = 9.81;

// Filled later require h_swl, it is b in the formulas
real_number B = 0.0;

// Constant used to define time integration
const real_number CFLnumber = 0.2;

// Minimum T
const real_number DtMin = 1.0e-9;

// Minimum Rho allowed
const real_number RhoMin = 700.0;

// Maximum Rho allowed
const real_number RhoMax = 1300.0;

// Filled in initialization
real_number max_fluid_height = 0.0;

// Properties

// FLUID or BOUNDARY
const size_t type = 0;

// Density
const int rho = 1;

// Density at step n-1
const int rho_prev = 2;

// Pressure
const int Pressure = 3;

// Delta rho calculated in the force calculation
const int drho = 4;

// calculated force
const int force = 5;

// velocity
const int velocity = 6;

// velocity at previous step
const int velocity_prev = 7;

const int red = 8;

const int red2 = 9;

// Type of the vector containing particles
typedef vector_dist_gpu< 3,
                         real_number,
                         aggregate< unsigned int,
                                    real_number,
                                    real_number,
                                    real_number,
                                    real_number,
                                    real_number[ 3 ],
                                    real_number[ 3 ],
                                    real_number[ 3 ],
                                    real_number,
                                    real_number > >
   particles;
//                                              |          |             |               |                |                | |
//                                              |               |            | |          |             |               | | | |
//                                              |               |            |
//                                             type      density       density        Pressure          delta            force
//                                             velocity        velocity        reduction     another
//                                                                     at n-1                           density at n - 1 buffer
//                                                                     reduction buffer

struct ModelCustom
{
   template< typename Decomposition, typename vector >
   inline void
   addComputation( Decomposition& dec, vector& distribtuedVector, size_t v, size_t p )
   {
      if( distribtuedVector.template getProp< type >( p ) == FLUID )
         dec.addComputationCost( v, 4 );
      else
         dec.addComputationCost( v, 3 );
   }

   template< typename Decomposition >
   inline void
   applyModel( Decomposition& dec, size_t v )
   {
      dec.setSubSubDomainComputationCost( v,
                                          dec.getSubSubDomainComputationCost( v ) * dec.getSubSubDomainComputationCost( v ) );
   }

   real_number
   distributionTol()
   {
      return 1.01;
   }
};

template< typename DistributedParticleVector >
__global__
void
equationOfState_kernel( DistributedParticleVector distributedVector, const RealType& B )
{
   auto i = GET_PARTICLE( distributedVector );

   const RealType rho_i = v( i );
   const RealType rho_frac = rho_a / rho0;
   p( i ) = B * ( std::pow( rho_frac, 7 ) - 1.f );
}

void
equationOfState( DistributedParticleVector& distributedVector )
{
   // particle iterator
   auto distributedParticleVectorGPUIterator = distributedVector.getDomainIteratorGPU();
   CUDA_LAUNCH( equationOfState_kernel, distributedParticleVectorGPUIterator, distributedVector.toKernel(), B );
}

__device__
__host__
RealType
smoothingFunction_W( const RealType& r, const RealType& h )
{
   const float wConst = 0.02611136f / ( h * h * h );  // 21/(16*PI*h^3)
   const float q = r / h;
   return wConst * ( 1.f + 2.f * q ) * ( 2.f - q ) * ( 2.f - q ) * ( 2.f - q ) * ( 2.f - q );
}

__device__
__host__
RealType
smoothingFunction_F( const RealType& r, const RealType& h )
{
   const float wConst = -0.2611136f / ( h * h * h * h * h );  // 21/(16*PI*h^5)*(5/8)
   const float q = r / h;
   return wConst * ( 2.f - q ) * ( 2.f - q ) * ( 2.f - q );
}

inline __device__
__host__
RealType
viscousTerm_Pi( const RealType& rhoI,
                const RealType& rhoJ,
                const RealType& drs,
                const RealType& drdv,
                const RealType& h,
                const RealType& alpha,
                const RealType& preventZeroEps )
{
   const RealType mu = h * drdv / ( drs * drs + preventZeroEps );
   return ( drdv < 0.f ) ? ( alpha * mu / ( rhoI + rhoJ ) ) : ( 0.f );
}

template< typename particles_type, typename fluid_ids_type, typename nearestNeighbors_type >
__global__
void
computeFluidInteractions_kernel( particles_type distribtuedVector,
                                 fluid_ids_type fids,
                                 nearestNeighbors_type nearestNeighbors,
                                 real_number W_dap,
                                 real_number cbar )
{
   unsigned int a;
   GET_PARTICLE_BY_ID( a, fids );

   // get properties of particle a
   const VectorType r_a = r( a );
   const unsigned int type_a = type( a );
   const RealType rho_a = rho( i );
   const RealType p_a = p( a );
   const VectorType v_a = v( a );

   VectorType dv_dt_a = { 0.f, 0.f, -gravity };
   RealType drho_dt_a = 0.f;
   RealType maxViscosity_a = 0.f;

   // get an iterator over the neighborhood particles of a
   auto neihgborParticlesIterator = nearestNeighbors.getnearestNeighborsIteratorBox( nearestNeighbors.getCell( r_a ) );

   // iterate over particles in neighborhood
   while( neihgborParticlesIterator.isNext() == true ) {
      const auto b = neihgborPartiicles.get_sort();
      const unsigned int type_b = type( b );
      if( type_a == BOUNDARY && type_b == BOUNDARY || a == b )  //TODO: It boundary/boundary check necessary?
      {
         ++neihgborParticlesIterator;
         continue;
      }

      const VectorType r_b = distributedVector.getPos( b );
      const VectorType v_b = v( b );
      const RealType p_b = p( b );
      const RealType rho_b = rho( b );

      const VectorType r_ab = xa - xb;
      const VectorType v_ab = va - vb;
      const RealType drs = norm( dr );

      if( drs < searchRadius )  //TODO: This condition is too deep
      {
         const VectorType gradW = r_ab * smoothingFunction_F( drs, h );
         const RealType pressureTerm = ( p_a + p_b ) / ( rho_a + rho_b );
         const RealType viscousTerm = viscousTerm_Pi( rho_i, rho_j, drs, ( r_ij, v_ij ), h, alpha, preventZeroEps );

         maxViscosity_a += viscousTerm;
         dv_dt_a += ( -1.f ) * ( pressureTerm + viscousTerm ) * DW * massb;
         drho_dt_a += ( v_ab, gradW ) * m;
      }

      ++neihgborParticlesIterator;
   }
   reductionBufferVisco( a ) = maxViscosity_a;
   dv_dt( a ) = dv_dt_a;
   drho_dt( a ) = drho_dt_a;
}

template< typename particles_type, typename fluid_ids_type, typename nearestNeighbors_type >
__global__
void
computeBoundaryInteractions_kernel( particles_type distribtuedVector,
                                    fluid_ids_type fbord,
                                    nearestNeighbors_type nearestNeighbors,
                                    real_number W_dap,
                                    real_number cbar )
{
   unsigned int a;
   GET_PARTICLE_BY_ID( a, fbord );

   // get properties of particle a
   const VectorType r_a = r( a );
   const unsigned int type_a = type( a );
   const VectorType v_a = v( a );

   RealType drho_dt_a = 0.f;
   RealType maxViscosity_a = 0.f;

   // get an iterator over the neighborhood particles of a
   auto neihgborParticlesIterator = nearestNeighbors.getnearestNeighborsIteratorBox( nearestNeighbors.getCell( r_a ) );

   // iterate over particles in neighborhood
   while( neihgborParticlesIterator.isNext() == true ) {
      const auto b = neihgborPartiicles.get_sort();
      const unsigned int type_b = type( b );
      if( type_a == BOUNDARY && type_b == BOUNDARY || a == b )  //TODO: Is boundary/boundary check necessary?
      {
         ++neihgborParticlesIterator;
         continue;
      }

      const VectorType r_b = distributedVector.getPos( b );
      const VectorType v_b = v( b );

      const VectorType r_ab = xa - xb;
      const VectorType v_ab = va - vb;
      const RealType drs = norm2( dr );

      if( drs < searchRadius )  //TODO: This condition is too deep
      {
         const VectorType gradW = r_ab * smoothingFunction_F( drs, h );
         drho_dt_a += ( v_ab, gradW ) * m;
      }
      ++neihgborParticlesIterator;
   }
   reductionBufferVisco( a ) = maxViscosity_a;
   dv_dt( a ) = dv_dt_a;
}

struct ChekFluidType
{
   __device__
   static bool
   check( int c )
   {
      return c == FLUID;
   }
};

struct CheckBoundaryType
{
   __device__
   static bool
   check( int c )
   {
      return c == BOUNDARY;
   }
};

template< typename CellList >
void
computeInteractions( DistributedParticleVector& distribtuedVector,
                     CellList& nearestNeighbors,
                     RealType& max_visc,
                     size_t cnt,
                     openfpm::vector_gpu< aggregate< int > >& fluid_ids,
                     openfpm::vector_gpu< aggregate< int > >& border_ids )
{
   // update the cell-list
   distributedVector.updateCellList< PARTICLE_TYPE, RHO, PRESSURE, VELOCITY >( nearestNeighbors );

   // get the particles fluid ids
   get_indexes_by_type< PARTICLE_TYPE, ChekFluidType >( distribtuedVector.getPropVectorSort(),
                                                        fluid_ids,
                                                        distribtuedVector.size_local(),
                                                        distribtuedVector.getVC().getGpuContext() );

   // get the particles fluid ids
   get_indexes_by_type< PARTICLE_TYPE, CheckBoundaryType >( distribtuedVector.getPropVectorSort(),
                                                            border_ids,
                                                            distribtuedVector.size_local(),
                                                            distribtuedVector.getVC().getGpuContext() );

   auto part = fluid_ids.getGPUIterator( 96 );
   CUDA_LAUNCH( computeFluidInteractions_kernel,
                part,
                distribtuedVector.toKernel_sorted(),
                fluid_ids.toKernel(),
                nearestNeighbors.toKernel(),
                W_dap,
                cbar );

   part = border_ids.getGPUIterator( 96 );
   CUDA_LAUNCH( computeBoundaryInteractions_kernel,
                part,
                distribtuedVector.toKernel_sorted(),
                border_ids.toKernel(),
                nearestNeighbors.toKernel(),
                W_dap,
                cbar );

   distribtuedVector.merge_sort< DV_DT, DRHO_DT, REDUCTION_REMOVE >( nearestNeighbors );
   max_visc = reduce_local< REDUCTION_REMOVE, _max_ >( distribtuedVector );  //TODO: _max_ is functional? Ugly name.
}

template< typename DistributedParticleVectorType >
__global__
void
getMaximumVelocityAndAcceleration_kernel( DistributedParticleVectorType distribtuedVector )
{
   auto a = GET_PARTICLE( distribtuedVector );
   reductionBufferRemove( a ) = norm( dv_dt( a ) ); //TODO: chose better reduction buffer name
   reductionBufferVisco( a ) = norm( v( a ) ); //TODO: chode better reduction buffer name
}

void
getMaximumVelocityAndAcceleration( DistributedParticleVector& distribtuedVector, RealType& maxAcc, RealType& maxVel )
{
   // particle iterator
   auto distributedParticleVectorGPUIterator = distributedVector.getDomainIteratorGPU();

   CUDA_LAUNCH( getMaximumVelocityAndAcceleration_kernel,
                distributedParticleVectorGPUIterator,
                distribtuedVector.toKernel() );

   maxAcc = reduce_local< REDUCTION_REMOVE, _max_ >( distribtuedVector ); //TODO: chose better reduction buffer name
   maxVel = reduce_local< REDUCTION_VISCO, _max_ >( distribtuedVector ); //TODO: chose better reduction buffer name

   Vcluster<>& vCluster = create_vcluster();
   vCluster.max( max_acc );
   vCluster.max( max_vel );
   vCluster.execute();
}

RealType
computeTimeStepSize( DistributedParticleVector& distribtuedVector, RealType viscDtMax )
{
   RealType maxAcc = 0.0;
   RealType maxVel = 0.0;
   getMaximumVelocityAndAcceleration( distribtuedVector, maxAcc, maxVel );

   // compute time step based on force per unit mass.
   const RealType dt_f = ( Maxacc ) ? sqrt( h / maxAcc ) : std::numeric_limits< float >::max();
   // compute time step based on CFL condition and maximum viscous forces.
   const real_number dt_cv = H / ( std::max( cbar, maxVel * 10.f ) + h * viscDtMax );
   // compute resulting time steop
   real_number dt = RealType( CFLnumber ) * std::min( dt_f, dt_cv );
   if( dt < RealType( dtMin ) )
      dt = RealType( dtMin );

   return dt;
}

template< typename DistributedParticleVector >
__global__
void
verletIntegrationScheme_kernel( DistributedParticleVector distributedVector,
                                const RealType dt,
                                const RealType dt2,
                                const RealType dt205 )
{
   auto i = GET_PARTICLE( distributedVector );

   // if the particle is boundary, update density
   if( type( i ) == ParticleTypes::Wall ) {
      v( i ) = 0.f;

      const RealType = rhoBackup_i = rho( i );
      const RealType = rhoNew_i = rho_old( i ) + dt2 * drho_dt( i );
      rho( i ) = ( rhoNew_i < rho0 ) ? rho0 : rhoNew_i;
      rho_old( i ) = rhoBackup_i;

      reductionBufferRemove( i ) = 0;
      return;
   }

   // if the particle is fluid, update position and density
   const VectorType dr_i = dt * v( i ) + dt205 * dv_dt( i );
   r( i ) += dr_i;
   const VectorType vToBackup_i = v( i );
   v( i ) = v_old( i ) + dt2 * dv_dt( i );
   const RealType rhoToBackup_i = rho( i );
   rho( i ) = rho_old( i ) + dt2 * drho_dt;

   // check if the particle go out of range in space and in density
   const VectorType r_i = r( i );
   const RealType rho_i = rho( i );
   if( r_i[ 0 ] < 0.0 || r_i[ 1 ] < 0.0 || r_i[ 2 ] < 0.0 || r_i[ 0 ] > 3.22 || r_i[ 1 ] > 1.0 || r_i[ 2 ] > 1.5
       || rho_i < RhoMin || rho_i > RhoMax ) {
      reductionBufferRemove( i ) = 1;
   }
   else {
      reductionBufferRemove( i ) = 0;
   }

   v_old( i ) = vToBackup_i;
   rho_old( i ) = rhoToBackup_i;
}

size_t cnt = 0;

void
verletIntegrationScheme( DistributedParticleVector& distributedVector, const RealType& dt )
{
   // particle iterator
   auto distributedParticleVectorGPUIterator = distributedVector.getDomainIteratorGPU();

   const RealType dt205 = dt * dt * 0.5;
   const RealType dt2 = dt * 2.0;

   CUDA_LAUNCH(
      verletIntegrationScheme_kernel, distributedParticleVectorGPUIterator, distributedVector.toKernel(), dt, dt2, dt205 );

   // remove the marked particles
   remove_marked< REDUCTION_REMOVE >( distributedVector );

   // increment the iteration counter
   cnt++;
}

template< typename DistributedParticleVector >
__global__
void
eulerIntegrationScheme_kernel( DistributedParticleVector& DistributedParticleVector, const RealType dt, const RealType dt205 )
{
   auto i = GET_PARTICLE( distributedVector );

   // if the particle is boundary, update density
   if( type( i ) == ParticleTypes::Wall ) {
      v( i ) = 0.f;

      const RealType = rhoBackup_i = rho( i );
      const RealType = rhoNew_i = rho( i ) + dt * drho_dt( i );
      rho( i ) = ( rhoNew_i < rho0 ) ? rho0 : rhoNew_i;
      rho_old( i ) = rhoBackup_i;

      reductionBufferRemove( i ) = 0;
      return;
   }

   // if the particle is fluid, update position and density
   const VectorType dr_i = dt * v( i ) + dt205 * dv_dt( i );
   r( i ) += dr_i;

   const VectorType vToBackup_i = v( i );
   v( i ) = v( i ) + dt * dv_dt( i );

   const RealType rhoToBackup_i = rho( i );
   rho( i ) = rho( i ) + dt * drho_dt( i );

   // check if the particle go out of range in space and in density
   const VectorType r_i = r( i );
   const RealType rho_i = rho( i );
   if( r_i[ 0 ] < 0.0 || r_i[ 1 ] < 0.0 || r_i[ 2 ] < 0.0 || r_i[ 0 ] > 3.22 || r_i[ 1 ] > 1.0 || r_i[ 2 ] > 1.5
       || rho_i < RhoMin || rho_i > RhoMax ) {
      reductionBufferRemove( i ) = 1;
   }
   else {
      reductionBufferRemove( i ) = 0;
   }

   v_old( i ) = vToBackup_i;
   rho_old( i ) = rhoToBackup_i;
}

void
eulerIntegrationScheme( DistributedParticleVector& distributedVector, const RealType& dt )
{
   // particle iterator
   auto distributedParticleVectorGPUIterator = distributedVector.getDomainIteratorGPU();

   const RealType dt205 = dt * dt * 0.5;

   CUDA_LAUNCH(
      eulerIntegrationScheme_kernel, distributedParticleVectorGPUIterator, DistributedParticleVector.toKernel(), dt, dt205 );

   // remove the particles
   remove_marked< REDUCTION_REMOVE >( distributedVector );

   // increment the iteration counter
   cnt++;
}

template< typename vector_type, typename nearestNeighbors_type >
__global__
void
sensor_pressure_gpu( vector_type distribtuedVector,
                     nearestNeighbors_type nearestNeighbors,
                     Point< 3, real_number > probe,
                     real_number* press_tmp )
{
   real_number tot_ker = 0.0;

   // Get the position of the probe i
   Point< 3, real_number > xp = probe;

   // get the iterator over the neighbohood particles of the probes position
   auto itg = nearestNeighbors.getnearestNeighborsIteratorBox( nearestNeighbors.getCell( xp ) );
   while( itg.isNext() ) {
      auto q = itg.get_sort();

      // Only the fluid particles are importants
      if( distribtuedVector.template getProp< type >( q ) != FLUID ) {
         ++itg;
         continue;
      }

      // Get the position of the neighborhood particle q
      Point< 3, real_number > xq = distribtuedVector.getPos( q );

      // Calculate the contribution of the particle to the pressure
      // of the probe
      real_number r = sqrt( norm2( xp - xq ) );

      real_number ker = Wab( r ) * ( MassFluid / rho_zero );

      // Also keep track of the calculation of the summed
      // kernel
      tot_ker += ker;

      // Add the total pressure contribution
      *press_tmp += distribtuedVector.template getProp< Pressure >( q ) * ker;

      // next neighborhood particle
      ++itg;
   }

   // We calculate the pressure normalizing the
   // sum over all kernels
   if( tot_ker == 0.0 ) {
      *press_tmp = 0.0;
   }
   else {
      *press_tmp = 1.0 / tot_ker * *press_tmp;
   }
}

template< typename Vector, typename CellList >
inline void
sensor_pressure( Vector& distribtuedVector,
                 CellList& nearestNeighbors,
                 openfpm::vector< openfpm::vector< real_number > >& press_t,
                 openfpm::vector< Point< 3, real_number > >& probes )
{
   Vcluster<>& vCluser = create_vcluster();

   press_t.add();

   for( size_t i = 0; i < probes.size(); i++ ) {
      // A float variable to calculate the pressure of the problem
      CudaMemory press_tmp_( sizeof( real_number ) );
      real_number press_tmp;

      // if the probe is inside the processor domain
      if( distribtuedVector.getDecomposition().isLocal( probes.get( i ) ) == true ) {
         CUDA_LAUNCH_DIM3( sensor_pressure_gpu,
                           1,
                           1,
                           distribtuedVector.toKernel_sorted(),
                           nearestNeighbors.toKernel(),
                           probes.get( i ),
                           (real_number*) press_tmp_.toKernel() );

         //distribtuedVector.merge<Pressure>(nearestNeighbors);

         // move calculated pressure on
         press_tmp_.deviceToHost();
         press_tmp = *(real_number*) press_tmp_.getPointer();
      }

      // This is not necessary in principle, but if you
      // want to make all processor aware of the history of the calculated
      // pressure we have to execute this
      vCluser.sum( press_tmp );
      vCluser.execute();

      // We add the calculated pressure into the history
      press_t.last().add( press_tmp );
   }
}

template< typename vector_type, typename nearestNeighbors_type >
__global__
void
sensor_water_level_gpu( vector_type distribtuedVector,
                        nearestNeighbors_type nearestNeighbors,
                        Point< 3, real_number > probe,
                        real_number* wl_tmp,
                        const size_t number_of_levels_to_test )
{
   unsigned int p = blockDim.x * blockIdx.x + threadIdx.x;
   if( p >= number_of_levels_to_test )
      return;

   real_number tot_ker = 0.0;

   // Get the position of the probe i
   Point< 3, real_number > level = { 0., 0., p * dp };
   Point< 3, real_number > xp = probe + level;

   // get the iterator over the neighbohood particles of the probes position
   auto itg = nearestNeighbors.getnearestNeighborsIteratorBox( nearestNeighbors.getCell( xp ) );
   while( itg.isNext() ) {
      auto q = itg.get_sort();

      // Only the fluid particles are important
      //if (distribtuedVector.template getProp<type>(q) != FLUID)
      //{
      //	++itg;
      //	continue;
      //}

      // Get the position of the neighborhood particle q
      Point< 3, real_number > xq = distribtuedVector.getPos( q );

      // Calculate the kernel occupancy
      real_number r = sqrt( norm2( xp - xq ) );
      real_number ker = Wab( r ) * ( MassFluid / distribtuedVector.template getProp< rho >( q ) );
      tot_ker += ker;

      // next neighborhood particle
      ++itg;
      //if( ker != 0.0 )
      //   printf(" << tot_ker, Wab, MassFluid, rho >>: [%f , %f, %f, %f]", tot_ker, ker, MassFluid, distribtuedVector.template
      //   getProp<rho>(q) );
   }
   wl_tmp[ p ] = tot_ker;
   //printf(" tot_ker: %f ", tot_ker);
}

template< typename Vector, typename CellList >
inline void
sensor_water_level( Vector& distribtuedVector,
                    CellList& nearestNeighbors,
                    openfpm::vector< openfpm::vector< real_number > >& water_level_t,
                    openfpm::vector< Point< 3, real_number > >& probes_water_level )
{
   Vcluster<>& vCluser = create_vcluster();

   water_level_t.add();

   for( size_t i = 0; i < probes_water_level.size(); i++ ) {
      //sampled water levels
      openfpm::vector< real_number > tested_water_levels;
      size_t number_of_levels_to_test = (size_t) std::floor( 1.5 / dp );  //TODO: approx. box height

      //A float variable to calculate the pressure of the problem
      CudaMemory wl_tmp_( number_of_levels_to_test * sizeof( real_number ) );
      real_number* wl_tmp;

      //if the probe is inside the processor domain
      if( distribtuedVector.getDecomposition().isLocal( probes_water_level.get( i ) ) == true ) {
         CUDA_LAUNCH_DIM3( sensor_water_level_gpu,
                           1,
                           number_of_levels_to_test,
                           distribtuedVector.toKernel_sorted(),
                           nearestNeighbors.toKernel(),
                           probes_water_level.get( i ),
                           (real_number*) wl_tmp_.toKernel(),
                           number_of_levels_to_test );

         //distribtuedVector.merge<Pressure>(nearestNeighbors);

         // move calculated pressure on
         wl_tmp_.deviceToHost();
         //wl_tmp = *(real_number *)wl_tmp_.getPointer();
         wl_tmp = static_cast< real_number* >( const_cast< void* >( wl_tmp_.getPointer() ) );
      }

      //Obtain the actual water level from the samples
      real_number water_level_temp = 0;
      //std::cout << "senzor: " << i << std::endl;
      //for( size_t j = 1 ; j < number_of_levels_to_test ; j ++)
      //{
      //   std::cout << wl_tmp[j] << " ";
      //   //if( wl_tmp[j] < 0.5f ){
      //   //   water_level_temp = j * dp;
      //   //   break;
      //   //}
      //}
      std::cout << std::endl;
      for( size_t j = 1; j < number_of_levels_to_test - 1; j++ ) {
         if( ( wl_tmp[ j ] < 0.5f ) && ( wl_tmp[ j + 1 ] < 0.5f ) ) {
            water_level_temp = j * dp;
            break;
         }
      }
      std::cout << "Water level temp = " << water_level_temp << std::endl;

      // This is not necessary in principle, but if you
      // want to make all processor aware of the history of the calculated
      // pressure we have to execute this
      vCluser.sum( water_level_temp );
      vCluser.execute();

      // We add the calculated pressure into the history
      water_level_t.last().add( water_level_temp );
   }
}

int
main( int argc, char* argv[] )
{
   // initialize the library
   openfpm_init( &argc, &argv );

   openfpm::vector_gpu< aggregate< int > > fluid_ids;
   openfpm::vector_gpu< aggregate< int > > border_ids;

   #ifdef CUDIFY_USE_CUDA
      cudaDeviceSetCacheConfig( cudaFuncCachePreferL1 );
   #endif

   // It contain for each time-step the value detected by the probes
   openfpm::vector< openfpm::vector< real_number > > press_t;
   openfpm::vector< Point< 3, real_number > > probes;
   std::vector< real_number > press_measured_times;

   openfpm::vector< openfpm::vector< real_number > > water_level_t;
   openfpm::vector< Point< 3, real_number > > probes_water_level;

   probes.add( { 0.8245, 0.471, 0.021 } );
   probes.add( { 0.8245, 0.471, 0.061 } );
   probes.add( { 0.8245, 0.471, 0.101 } );
   probes.add( { 0.8245, 0.471, 0.141 } );
   probes.add( { 0.8035, 0.471, 0.1645 } );
   probes.add( { 0.7635, 0.471, 0.1645 } );
   probes.add( { 0.7235, 0.471, 0.1645 } );
   probes.add( { 0.6835, 0.471, 0.1645 } );

   probes_water_level.add( { 0.496, 0.5, 0.0 } );
   probes_water_level.add( { 0.992, 0.5, 0.0 } );
   probes_water_level.add( { 1.488, 0.5, 0.0 } );
   probes_water_level.add( { 2.638, 0.5, 0.0 } );

   // Here we define our domain a 2D box with internals from 0 to 1.0 for x and y
   //Box<3,real_number> domain({-0.05,-0.05,-0.05},{1.7010,0.7065,0.511});
   //size_t sz[3] = {413,179,133};
   Box< 3, real_number > domain( { -0.05, -0.05, -0.05 }, { 3.3210, 1.1065, 1.511 } );
   //TOOD: Parametrize this
   size_t sz_x = (size_t) std::ceil( ( 3.3210 + 0.05 ) / dp );
   size_t sz_y = (size_t) std::ceil( ( 1.1065 + 0.05 ) / dp );
   size_t sz_z = (size_t) std::ceil( ( 1.511 + 0.05 ) / dp );
   size_t sz[ 3 ] = { sz_x, sz_y, sz_z };

   // Fill W_dap
   W_dap = 1.0 / Wab( H / 1.5 );

   // Here we define the boundary conditions of our problem
   size_t bc[ 3 ] = { NON_PERIODIC, NON_PERIODIC, NON_PERIODIC };

   // extended boundary around the domain, and the processor domain
   Ghost< 3, real_number > g( 2 * H );

   particles distribtuedVector( 0, domain, bc, g, DEC_GRAN( 128 ) );

   //! \cond [draw fluid] \endcond

   // You can ignore all these dp/2.0 is a trick to reach the same initialization
   // of Dual-SPH that use a different criteria to draw particles
   //Box<3,real_number> fluid_box({dp/2.0,dp/2.0,dp/2.0},{0.4+dp/2.0,0.67-dp/2.0,0.3+dp/2.0});
   Box< 3, real_number > fluid_box( { 1.992 + dp / 2.0, dp / 2.0, dp / 2.0 },
                                    { 3.22 - dp / 2.0, 1.0 - dp / 2.0, 0.55 + dp / 2.0 } );

   // return an iterator to the fluid particles to add to distribtuedVector
   auto fluid_it = DrawParticles::DrawBox( distribtuedVector, sz, domain, fluid_box );

   // here we fill some of the constants needed by the simulation
   max_fluid_height = fluid_it.getBoxMargins().getHigh( 2 );
   h_swl = fluid_it.getBoxMargins().getHigh( 2 ) - fluid_it.getBoxMargins().getLow( 2 );
   B = ( coeff_sound ) * (coeff_sound) *gravity * h_swl * rho_zero / gamma_;
   cbar = coeff_sound * sqrt( gravity * h_swl );

   // for each particle inside the fluid box ...
   while( fluid_it.isNext() ) {
      // ... add a particle ...
      distribtuedVector.add();

      // ... and set it position ...
      distribtuedVector.getLastPos()[ 0 ] = fluid_it.get().get( 0 );
      distribtuedVector.getLastPos()[ 1 ] = fluid_it.get().get( 1 );
      distribtuedVector.getLastPos()[ 2 ] = fluid_it.get().get( 2 );

      // and its type.
      distribtuedVector.template getLastProp< type >() = FLUID;

      // We also initialize the density of the particle and the hydro-static pressure given by
      //
      // rho_zero*g*h = P
      //
      // rho_p = (P/B + 1)^(1/Gamma) * rho_zero
      //

      distribtuedVector.template getLastProp< Pressure >() =
         rho_zero * gravity * ( max_fluid_height - fluid_it.get().get( 2 ) );

      distribtuedVector.template getLastProp< rho >() =
         pow( distribtuedVector.template getLastProp< Pressure >() / B + 1, 1.0 / gamma_ ) * rho_zero;
      distribtuedVector.template getLastProp< rho_prev >() = distribtuedVector.template getLastProp< rho >();
      distribtuedVector.template getLastProp< velocity >()[ 0 ] = 0.0;
      distribtuedVector.template getLastProp< velocity >()[ 1 ] = 0.0;
      distribtuedVector.template getLastProp< velocity >()[ 2 ] = 0.0;

      distribtuedVector.template getLastProp< velocity_prev >()[ 0 ] = 0.0;
      distribtuedVector.template getLastProp< velocity_prev >()[ 1 ] = 0.0;
      distribtuedVector.template getLastProp< velocity_prev >()[ 2 ] = 0.0;

      // next fluid particle
      ++fluid_it;
   }

   // recipient
   Box< 3, real_number > recipient1( { 0.0, 0.0, 0.0 }, { 3.22 + dp / 2.0, 1.0 + dp / 2.0, 1.0 + dp / 2.0 } );
   Box< 3, real_number > recipient2( { dp, dp, dp }, { 3.22 - dp / 2.0, 1.0 - dp / 2.0, 1.0 + dp / 2.0 } );


   // obstacle
   Box< 3, real_number > obstacle1( { 0.66, 0.3 - dp / 2.0, 0.0 }, { 0.82 + dp / 2.0, 0.7, 0.16 + dp / 2.0 } );
   Box< 3, real_number > obstacle2( { 0.66 + dp, 0.3 + dp / 2.0, 0.0 }, { 0.82 - dp / 2.0, 0.7 - dp, 0.16 - dp / 2.0 } );
   Box< 3, real_number > obstacle3( { 0.66 + dp, 0.3, 0.0 }, { 1.02, 0.7, 0.16 } );

   openfpm::vector< Box< 3, real_number > > holes;
   holes.add( recipient2 );
   holes.add( obstacle1 );

   auto bound_box = DrawParticles::DrawSkin( distribtuedVector, sz, domain, holes, recipient1 );
   while( bound_box.isNext() ) {
      distribtuedVector.add();
      distribtuedVector.getLastPos()[ 0 ] = bound_box.get().get( 0 );
      distribtuedVector.getLastPos()[ 1 ] = bound_box.get().get( 1 );
      distribtuedVector.getLastPos()[ 2 ] = bound_box.get().get( 2 );
      distribtuedVector.template getLastProp< type >() = BOUNDARY;
      distribtuedVector.template getLastProp< rho >() = rho_zero;
      distribtuedVector.template getLastProp< rho_prev >() = rho_zero;
      distribtuedVector.template getLastProp< velocity >()[ 0 ] = 0.0;
      distribtuedVector.template getLastProp< velocity >()[ 1 ] = 0.0;
      distribtuedVector.template getLastProp< velocity >()[ 2 ] = 0.0;
      distribtuedVector.template getLastProp< velocity_prev >()[ 0 ] = 0.0;
      distribtuedVector.template getLastProp< velocity_prev >()[ 1 ] = 0.0;
      distribtuedVector.template getLastProp< velocity_prev >()[ 2 ] = 0.0;
      ++bound_box;
   }

   auto obstacle_box = DrawParticles::DrawSkin( distribtuedVector, sz, domain, obstacle2, obstacle1 );
   while( obstacle_box.isNext() ) {
      distribtuedVector.add();
      distribtuedVector.getLastPos()[ 0 ] = obstacle_box.get().get( 0 );
      distribtuedVector.getLastPos()[ 1 ] = obstacle_box.get().get( 1 );
      distribtuedVector.getLastPos()[ 2 ] = obstacle_box.get().get( 2 );
      distribtuedVector.template getLastProp< type >() = BOUNDARY;
      distribtuedVector.template getLastProp< rho >() = rho0;
      distribtuedVector.template getLastProp< rho_prev >() = rho0;
      distribtuedVector.template getLastProp< velocity >()[ 0 ] = 0.0;
      distribtuedVector.template getLastProp< velocity >()[ 1 ] = 0.0;
      distribtuedVector.template getLastProp< velocity >()[ 2 ] = 0.0;
      distribtuedVector.template getLastProp< velocity_prev >()[ 0 ] = 0.0;
      distribtuedVector.template getLastProp< velocity_prev >()[ 1 ] = 0.0;
      distribtuedVector.template getLastProp< velocity_prev >()[ 2 ] = 0.0;
      ++obstacle_box;
   }

   distribtuedVector.map();

   // Now that we fill the vector with particles
   ModelCustom modelCustom;
   distribtuedVector.addComputationCosts( modelCustom );
   distribtuedVector.getDecomposition().decompose();
   distribtuedVector.map();

   // initialization is done on CPU on GPU we are doing the main loop, so first we offload all properties on GPU
   distribtuedVector.hostToDevicePos();
   distribtuedVector.template hostToDeviceProp< TYPE, RHO, RHO_PREV, PRESSURE, VELOCITY >();
   distribtuedVector.ghost_get< type, rho, Pressure, velocity >( RUN_ON_DEVICE );

   // initialize neighbor search structures
   auto nearestNeighbors = distribtuedVector.getCellListGPU /*<CELLLIST_GPU_SPARSE<3,float>>*/ ( searchRadius );
   nearestNeighbors.setBoxnearestNeighbors( 2 );

   // added timers to track every operation inside the time loop
   timer timerSimulationTotal;
   timer timerVCluster;
   float timerVClusterTotal = 0.f;
   timer timer_interaction;
   float interaction_total_time = 0.f;
   timer timerPressure;
   float timerPressureTotal = 0.f;
   timer timerIntegration;
   float timerIntegrationTotal = 0.f;
   timer timerRebalancing;
   float timerRebalancingTotal = 0.f;
   timer timerMap;
   float map_total_time = 0.f;
   timer timerComputeTimeStep;
   float timerComputeTimeStepTotal = 0.f;
   timer timer_ghosts;
   float ghost_total_time = 0.f;

   // sutaks timers,simulation time and step
   int simulationStep = 0;
   float time = 0.0;
   size_t counterWrite = 0;
   size_t counterIntegrationScheme = 0;
   size_t conouterRebalancing = 0;
   float counterSensors = 0;
   float writePeriodSensors = 0.01;

   timerSimulationTotal.start();

   while( t <= simulatioEndTime ) {
      simulationStep++;

      timerVCluster.start();
      Vcluster<>& vCluser = create_vcluster();
      timerVCluster.stop();
      timerVClusterTotal += timerVCluster.getwct();
      timer it_time;
      it_time.start();

      // every 300 steps, perform domain rebbalancing
      timerRebalancing.start();
      conouterRebalancing++;
      if( conouterRebalancing == 300 ) {
         distribtuedVector.map( RUN_ON_DEVICE );

         // eebalancer works on CPU, so move data to CPU
         distribtuedVector.deviceToHostPos();
         distribtuedVector.template deviceToHostProp< PARTICLE_TYPE >();

         conouterRebalancing = 0;
         ModelCustom modelCustom;
         distribtuedVector.addComputationCosts( modelCustom );
         distribtuedVector.getDecomposition().decompose();

         if( vCluser.getProcessUnitID() == 0 ) {
            std::cout << "REBALANCED " << conouterRebalancing << std::endl;
         }
      }
      timerRebalancing.stop();
      timerRebalancingTotal += timerRebalancing.getwct();

      timerMap.start();
      distribtuedVector.map( RUN_ON_DEVICE );
      timerMap.stop();
      timerRebalancingTotal += timerMap.getwct();

      // calculate pressure from the density
      timerPressure.start();
      equationOfState( distribtuedVector );
      timerPressure.stop();
      timerPressureTotal += timerPressure.getwct();

      timer_ghosts.start();
      distribtuedVector.ghost_get< PARTICLE_TYPE, RHO, PRESSURE, VELOCITY >( RUN_ON_DEVICE );
      timer_ghosts.stop();
      ghost_total_time += timer_ghosts.getwct();

      // calculate forces
      RealType maxViscosity = 0.0;
      timer_interaction.start();
      computeInteractions( distribtuedVector, nearestNeighbors, maxViscosity, cnt, fluid_ids, border_ids );
      timer_interaction.stop();
      interaction_total_time += timer_interaction.getwct();

      timerComputeTimeStep.start();
      // get maximum of viscosity term across processors
      vCluser.max( maxViscosity );
      vCluser.execute();
      // calculate possible time steop
      const RealType dt = computeTimeStepSize( distribtuedVector, maxViscosity );
      timerComputeTimeStep.stop();
      timerComputeTimeStepTotal += timerComputeTimeStep.getwct();

      // perform step with integration scheme
      timerIntegration.start();
      counterIntegrationScheme++;
      if( counterIntegrationScheme < 40 )
         verletIntegrationScheme( distribtuedVector, dt );
      else {
         eulerIntegrationScheme( distribtuedVector, dt );
         counterIntegrationScheme = 0;
      }
      timerIntegration.stop();
      timerIntegrationTotal += timerIntegration.getwct();

      time += dt;

      if( counterSensors <= time ) {
         // sensor pressure require update ghost, so we ensure that particles are distributed correctly and ghost are updated
         distribtuedVector.map( RUN_ON_DEVICE );
         distribtuedVector.ghost_get< PARTICLE_TYPE, RHO, PRESSURE, VELOCITY >( RUN_ON_DEVICE );
         distribtuedVector.updateCellList( nearestNeighbors );

         // calculate the pressure at the sensor points
         sensor_pressure( distribtuedVector, nearestNeighbors, press_t, probes );
         sensor_water_level( distribtuedVector, nearestNeighbors, water_level_t, probes_water_level );
         press_measured_times.push_back( time );
         counterSensors += writePeriodSensors;
      }

      if( counterWrite < time * 10 ) {
         std::cout << "Writing output in time:  " << time << std::endl;
         // Sensor pressure require update ghost, so we ensure that particles are distributed correctly and ghost are updated
         // NOTE: I don't think this is necessary for output
         distribtuedVector.map( RUN_ON_DEVICE );
         distribtuedVector.ghost_get< type, rho, Pressure, velocity >( RUN_ON_DEVICE );
         distribtuedVector.updateCellList( nearestNeighbors );

         // to write out the particles, move all data back to CPU
         distribtuedVector.deviceToHostPos();
         distribtuedVector.deviceToHostProp< TYPE,
                                             RHO,
                                             RHO_OLD,
                                             PRESSURE,
                                             DRHO_DT,
                                             DV_DT,
                                             VELOCITY,
                                             VELOCITY_OLD,
                                             REDUCTION_REMOVE,
                                             REDUCTION_VISCO >();

         // We copy on another vector with less properties to reduce the size of the output
         vector_dist_gpu< 3,
                          RealType,
                          aggregate< unsigned int, VectorType > > distribtuedVector_out( distribtuedVector.getDecomposition(), 0 );
         auto distributedParticleVectorIterator = distribtuedVector.getDomainIterator();

         while( distributedParticleVectorIterator.isNext() ) {
            auto p = distributedParticleVectorIterator.get();

            distribtuedVector_out.add();
            const VectorType r_p = r( p ); //TODO: The position interface is not vectorized
            distribtuedVector_out.getLastPos()[ 0 ] = r_p[ 0 ];
            distribtuedVector_out.getLastPos()[ 1 ] = r_p[ 1 ];
            distribtuedVector_out.getLastPos()[ 2 ] = r_p[ 2 ];
            distribtuedVector_out.template getLastProp< 0 >() = type( p );
            distribtuedVector_out.template getLastProp< 1 >() = v( p );

            ++distributedParticleVectorIterator;
         }

         distribtuedVector_out.write_frame( "Particles", write, VTK_WRITER | FORMAT_BINARY );
         counterWrite++;

         if( vCluser.getProcessUnitID() == 0 ) {
            std::cout << "TIME: " << t << "  write " << it_time.getwct() << "   " << conouterRebalancing << "   " << cnt
                      << " Max visc: " << max_visc << "   " << distribtuedVector.size_local() << std::endl;
         }
      }

      if( vCluser.getProcessUnitID() == 0 ) {
         std::cout << "TIME: " << t << "  " << it_time.getwct() << "   " << conouterRebalancing << "   " << cnt << " Max visc: " << max_visc
                   << "   " << distribtuedVector.size_local() << std::endl;
      }
   }

   timerSimulationTotal.stop();

   std::cout << "TIME MEASUREMENT RESULTS:" << std::endl;
   std::cout << "Time to complete: " << timerSimulationTotal.getwct() << " seconds" << std::endl;
   std::cout << "Vcluster: " << timerVClusterTotal << " seconds" << std::endl;
   std::cout << "Interaction: " << interaction_total_time << " seconds" << std::endl;
   std::cout << "Pressure: " << timerPressureTotal << " seconds" << std::endl;
   std::cout << "Integration: " << timerIntegrationTotal << " seconds" << std::endl;
   std::cout << "Reabalancing: " << timerRebalancingTotal << " seconds" << std::endl;
   std::cout << "Map: " << map_total_time << " seconds" << std::endl;
   std::cout << "Reduction: " << timerComputeTimeStepTotal << " seconds" << std::endl;
   std::cout << "Ghost: " << ghost_total_time << " seconds" << std::endl;
   std::cout << "Number of steps: " << simulationStep << " seconds" << std::endl;

   //Write pressure sensor outputs
   std::ofstream file_probes;
   file_probes.open( "pressure_sensors.csv" );
   for( size_t i = 0; i < press_measured_times.size(); i++ ) {
      file_probes << press_measured_times[ i ];
      for( size_t j = 0; j < probes.size(); j++ ) {
         file_probes << " " << press_t.get( i ).get( j );
      }
      file_probes << std::endl;
   }
   file_probes.close();

   //Write pressure sensor outputs
   std::ofstream file_probes_wl;
   file_probes_wl.open( "pressure_waterLevel.csv" );
   for( size_t i = 0; i < press_measured_times.size(); i++ ) {
      file_probes_wl << press_measured_times[ i ];
      for( size_t j = 0; j < probes_water_level.size(); j++ ) {
         file_probes_wl << " " << water_level_t.get( i ).get( j );
      }
      file_probes_wl << std::endl;
   }
   file_probes_wl.close();

   //Write timer into json like structure
   std::ofstream file_timers;
   file_timers.open( "timers.json" );

   file_timers << "{" << std::endl;
   file_timers << "	\"integrate\": \"" << timerIntegrationTotal << "\"," << std::endl;
   file_timers << "	\"integrate-average\": \"" << timerIntegrationTotal / simulationStep << "\"," << std::endl;
   file_timers << "	\"interaction\": \"" << interaction_total_time << "\"," << std::endl;
   file_timers << "	\"interaction-average\": \"" << interaction_total_time / simulationStep << "\"," << std::endl;
   file_timers << "	\"pressure-update\": \"" << timerPressureTotal << "\"," << std::endl;
   file_timers << "	\"pressure-update-average\": \"" << timerPressureTotal / simulationStep << "\"," << std::endl;
   file_timers << "	\"vcluster\": \"" << timerVClusterTotal << "\"," << std::endl;
   file_timers << "	\"vcluster-average\": \"" << timerVClusterTotal / simulationStep << "\"," << std::endl;
   file_timers << "	\"rebalancing\": \"" << timerRebalancingTotal << "\"," << std::endl;
   file_timers << "	\"rebalancing-average\": \"" << timerRebalancingTotal / simulationStep << "\"," << std::endl;
   file_timers << "	\"map\": \"" << map_total_time << "\"," << std::endl;
   file_timers << "	\"map-average\": \"" << map_total_time / simulationStep << "\"," << std::endl;
   file_timers << "	\"reduction\": \"" << timerComputeTimeStepTotal << "\"," << std::endl;
   file_timers << "	\"timerComputeTimeStepTotal-average\": \"" << timerComputeTimeStepTotal / simulationStep << "\"," << std::endl;
   file_timers << "	\"ghost\": \"" << ghost_total_time << "\"," << std::endl;
   file_timers << "	\"ghost-average\": \"" << ghost_total_time / simulationStep << "\"," << std::endl;
   file_timers << "	\"total\": \"" << timerSimulationTotal.getwct() << "\"," << std::endl;
   file_timers << "	\"total-average\": \"" << timerSimulationTotal.getwct() / simulationStep << "\"" << std::endl;
   file_timers << "}" << std::endl;

   file_timers.close();

   openfpm_finalize();
}

#else

int
main( int argc, char* argv[] )
{
   return 0;
}

#endif
